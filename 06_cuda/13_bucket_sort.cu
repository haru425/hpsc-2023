
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <vector>


__global__ void add(int* d_bucket, int* d_key, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        atomicAdd(&d_bucket[d_key[i]], 1);
    }
}

__global__ void sort(int* d_bucket, int* d_key, int n, int range) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = 0;
    for (int k=0; k<range; k++) {
        for (int l=0; l<d_bucket[k]; l++) {
            if (i == j) {
                d_key[i] = k;
            }
            j++;
        }
    }
}


int main() {
  int n = 50;
  int range = 5;

  int *d_key;
  hipMallocManaged(&d_key, n*sizeof(int));

  for (int i=0; i<n; i++) {
    d_key[i] = rand() % range;
    printf("%d ", d_key[i]);
  }
  printf("\n");

  int *d_bucket;
  
  hipMallocManaged(&d_bucket, n*sizeof(int));
  for (int i=0; i<range; i++) {
    d_bucket[i] = 0;
  }

  add<<<1, n>>>(d_bucket, d_key, n);
  hipDeviceSynchronize();

  sort<<<1, n>>>(d_bucket, d_key, n, range);
  hipDeviceSynchronize();
  
  for (int i=0; i<n; i++) {
    printf("%d ",d_key[i]);
  }
  printf("\n");
  hipFree(d_bucket);
  hipFree(d_key);
}
